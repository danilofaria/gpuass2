#ifndef __CUDACC__ 
#define __CUDACC__
#endif

#include <stdio.h>     
#include <stdlib.h>     
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <sstream>
#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include <stdexcept>      // std::invalid_argument
#include <assert.h>     /* assert */






#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

// error checking for CUDA calls: use this around ALL your calls!
#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
 
// cannot return values, so must store
// result in global memory location ("count")
// also: must make sure this thread maps to useful data! (what if
// the # of threads is > than the number of data elements!)
__global__ void histogram_gpu (unsigned int max, unsigned int *A, unsigned int *histogram)
{
 
    int n = blockDim.x * blockIdx.x + threadIdx.x;
 
    // do nothing if we are not in the useable space of
    // threads (see kernel launch call: you may be creating
    // more threads than you need)
    if (n >= max) return;
 
    unsigned int a = A[n];
 
    atomicAdd(&histogram[(a-1)/100], 1);
 
}
 

__global__ void histogram_gpu2 (unsigned int max, unsigned int *A, unsigned int *histogram)
{
 
    __shared__ unsigned int s_histogram [10];
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    int tId = threadIdx.x;
    if (tId < 10) 
        s_histogram[tId]=0;
    __syncthreads();
 
    // do nothing if we are not in the useable space of
    // threads (see kernel launch call: you may be creating
    // more threads than you need)
    if (n >= max) return;
 
    unsigned int a = A[n];
 
    atomicAdd(&s_histogram[(a-1)/100], 1);
    __syncthreads();

    if (tId < 10) 
        histogram[tId] = s_histogram[tId];
 
}

int main (int argc, char *argv[])
{
 
    if (argc <= 1) {
        printf("please pass in the name of a file\n");
        return 0;
    }

    std::ifstream infile;
    char * filename = argv[1];
    infile.open(filename);

    if (!infile) {
        printf("please indicate a valid filename\n");
        return 0;
    }

    int x, y, z;
    infile >> x >> y >> z;
    int maxTested = x * y *z;
    unsigned int n, i;


    // std::vector<unsigned int> intVec;
    // while(infile >> n) intVec.push_back(n);
    // how many test do we wish to make:
    // unsigned int maxTested = intVec.size(); 

    struct timeval t0, t1, t2;
 
 
    // allocate the array of integers to hold the data:
    unsigned int *h_intAArray;
    h_intAArray = (unsigned int *) malloc (maxTested * sizeof (unsigned int));
 

    std::string line;
    i =0;
    while(getline(infile,line)){
        std::stringstream   linestream(line);
        std::string         value;
        while(getline(linestream,value,','))
        {
            char * pEnd;
            int n = std::strtol (value.c_str(),&pEnd,10);
            if(n==0){
                continue;
            }else{
            h_intAArray[i] = n;
            i++;}
        }
        // Line Finished
    }
    assert (maxTested==i);

    // // fill it with numbers in file
    // for (unsigned int i = 0; i < maxTested; ++i) {
    //     infile >> n;
    //     h_intAArray[i] = n;
    // }

    // start basic timing:
    gettimeofday (&t0, 0);


    int histogram[10];
    for(int i=0;i<10;i++) histogram[i] = 0;
    // count how many are prime:
    for (int i = 0; i < maxTested; ++i) {
        n = h_intAArray[i];
        histogram[(n-1)/100]++; 
    }

    // how much time has elapsed?
    gettimeofday (&t1, 0);
 
    //
    // GPU version
    //
 
    // allocate the A array on the GPU, and copy the data over:
    unsigned int *d_intAArray;
    // allocate the histogram array on the GPU
    unsigned int *d_histogram;
 
    GPU_CHECKERROR(
    hipMalloc ((void **) &d_intAArray, maxTested * sizeof (unsigned int))
    );
 
    GPU_CHECKERROR(
    hipMemcpy ((void *) d_intAArray,
                (void *) h_intAArray,
                maxTested * sizeof (unsigned int),
                hipMemcpyHostToDevice)
    );

    GPU_CHECKERROR(
    hipMalloc ((void **) &d_histogram, 10 * sizeof (unsigned int))
    );
 
    GPU_CHECKERROR(
        hipMemset ((void *) d_histogram, 0, 10 * sizeof (unsigned int))
    );
 
    // we want to run a grid of 512-thread blocks (for reasons you
    // will understand later. How many such blocks will we need?
    // NOTE: be SURE to prevent integer division if you use this
    // snippet: that "1.0*" is absolutely required to prevent
    // rounding before the ceil() call:
    unsigned int threads_per_block = 512;
    unsigned int num_blocks = ceil (maxTested / (1.0*threads_per_block) );
 
    // launch the kernel:
    histogram_gpu<<<num_blocks, threads_per_block>>>
                                        (maxTested,
                                        d_intAArray,
                                        d_histogram);
 
    // get back the histogram:
    unsigned int h_histogram[10];
 
    hipMemcpy ((void *) h_histogram,
                (void *) d_histogram,
                10 * sizeof(unsigned int),
                hipMemcpyDeviceToHost);
    
    // make sure the GPU is finished doing everything!
    hipDeviceSynchronize();

    // finish timing:
    gettimeofday (&t2, 0);
 
    // free up the memory:
    hipFree (d_intAArray);
    hipFree (d_histogram);
    free (h_intAArray);
 
    // complete the timing:
    float timdiff1 = (1000000.0*(t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)) / 1000000.0;
    float timdiff2 = (1000000.0*(t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec)) / 1000000.0;

    printf ("%.2f\n", timdiff1);
    for(int i=0;i<10;i++){
        printf ("%d ", histogram[i]);
    }
    printf ("\n");

    printf ("%.2f\n", timdiff2);
    for(int i=0;i<10;i++){
        printf ("%d ", h_histogram[i]);
    }
    printf ("\n");
 
    // printf ("%d %.2f %d %.2f\n", primeCount, timdiff1, h_numPrimes, timdiff2);
  
}