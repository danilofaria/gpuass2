#include "hip/hip_runtime.h"
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <stdio.h>     
#include <stdlib.h>     
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <sstream>
#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include <stdexcept>      // std::invalid_argument
#include <assert.h>     /* assert */



#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

// error checking for CUDA calls: use this around ALL your calls!
#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
 
 __global__ void histogram_gpu (unsigned int x_dim, unsigned int y_dim, unsigned int z_dim, unsigned int *A, unsigned int *histogram)
{
    unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int z = blockDim.z * blockIdx.z + threadIdx.z;
    // do nothing if we are not in the useable space of
    // threads (see kernel launch call: you may be creating
    // more threads than you need)
    if (x >= x_dim || y >= y_dim || z >= z_dim) return;

    unsigned int n = x + y * x_dim + z * (x_dim*y_dim);
 
    unsigned int a = A[n];

    atomicAdd(&histogram[(a-1)/100], 1);
 
}

__global__ void histogram_gpu2 (unsigned int x_dim, unsigned int y_dim, unsigned int z_dim, unsigned int *A, unsigned int *histogram)
{
 
    __shared__ unsigned int s_histogram [10];
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int z = blockDim.z * blockIdx.z + threadIdx.z;
    int tId = threadIdx.x;
    bool smallBlock = (blockDim.x < 10);

    if(smallBlock)
        if (threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0)
        for(int i = 0; i < 10; i++) s_histogram[i]=0;
    else
        s_histogram[tId]=0;

    __syncthreads();
 
    // do nothing if we are not in the useable space of
    // threads (see kernel launch call: you may be creating
    // more threads than you need)
    if (x >= x_dim || y >= y_dim || z >= z_dim) return;
    unsigned int n = x + y * x_dim + z * (x_dim*y_dim);

    unsigned int a = A[n];
 
    atomicAdd(&s_histogram[(a-1)/100], 1);
    __syncthreads();

    if (smallBlock){
        if (threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0)
        for(int i = 0; i < 10; i++) 
            atomicAdd(&histogram[i], s_histogram[i]); 
    }
    else 
        atomicAdd(&histogram[tId], s_histogram[tId]); 
}

int main (int argc, char *argv[])
{
 
    if (argc <= 1) {
        printf("please pass in the name of a file\n");
        return 0;
    }

    std::ifstream infile;
    char * filename = argv[1];
    infile.open(filename);

    if (!infile) {
        printf("please indicate a valid filename\n");
        return 0;
    }

    unsigned int x, y, z;
    infile >> x >> y >> z;
    int maxTested = x * y *z;
    unsigned int i;


    // std::vector<unsigned int> intVec;
    // while(infile >> n) intVec.push_back(n);
    // how many test do we wish to make:
    // unsigned int maxTested = intVec.size(); 

    struct timeval t0, t1, t2, t3;
 
 
    // allocate the array of integers to hold the data:
    unsigned int *h_intAArray;
    h_intAArray = (unsigned int *) malloc (maxTested * sizeof (unsigned int));
 

    std::string line;
    i =0;
    while(getline(infile,line)){
        std::stringstream   linestream(line);
        std::string         value;
        while(getline(linestream,value,','))
        {
            char * pEnd;
            int n = std::strtol (value.c_str(),&pEnd,10);
            if(n==0){
                continue;
            }else{
            h_intAArray[i] = n;
            i++;}
        }
        // Line Finished
    }
    assert (maxTested==i);

    // printf ("x %d, y %d, z %d, maxTested %d\n", x,y,z,maxTested);

    // CPU
    // unsigned int n;
    // unsigned int histogram[10];
    // for(int i=0;i<10;i++) histogram[i] = 0;
    // // count how many are prime:
    // for (int i = 0; i < maxTested; ++i) {
    //     n = h_intAArray[i];
    //     histogram[(n-1)/100]++; 
    // }

 
    //
    // GPU version
    //
 
    // allocate the A array on the GPU, and copy the data over:
    unsigned int *d_intAArray;
    // allocate the histograms array on the GPU
    unsigned int *d_histogram1;
    unsigned int *d_histogram2;
 
    GPU_CHECKERROR(
    hipMalloc ((void **) &d_intAArray, maxTested * sizeof (unsigned int))
    );
 
    GPU_CHECKERROR(
    hipMemcpy ((void *) d_intAArray,
                (void *) h_intAArray,
                maxTested * sizeof (unsigned int),
                hipMemcpyHostToDevice)
    );

    GPU_CHECKERROR(
    hipMalloc ((void **) &d_histogram1, 10 * sizeof (unsigned int))
    );
 
    GPU_CHECKERROR(
        hipMemset ((void *) d_histogram1, 0, 10 * sizeof (unsigned int))
    );    
 
    // we want to run a grid of 512-thread blocks (for reasons you
    // will understand later. How many such blocks will we need?
    // NOTE: be SURE to prevent integer division if you use this
    // snippet: that "1.0*" is absolutely required to prevent
    // rounding before the ceil() call:
    unsigned int threads_per_block = 512;
    unsigned int num_blocks = ceil (maxTested / (1.0*threads_per_block) );

    int x_dim = min(8,x), y_dim = min(8,y), z_dim = min(8,z);

    unsigned int num_blocks_x = ceil (1.0*x / (1.0*x_dim) );
    unsigned int num_blocks_y = ceil (1.0*y / (1.0*y_dim) );
    unsigned int num_blocks_z = ceil (1.0*z / (1.0*z_dim) );

    // printf ("x_dim %d, y_dim %d, z_dim %d \n", x_dim,y_dim,z_dim);
    // printf ("num_blocks_x %d, num_blocks_y %d, num_blocks_z %d \n", num_blocks_x,num_blocks_y,num_blocks_z);

    // start basic timing:
    gettimeofday (&t0, 0);

    dim3 dimGrid(num_blocks_x, num_blocks_y, num_blocks_z);
    dim3 dimBlock(x_dim, y_dim, z_dim); 

    // launch the kernel (without shared memory):
    histogram_gpu<<<dimGrid, dimBlock>>>
                                        (x,y,z,
                                        d_intAArray,
                                        d_histogram1);
 
    // get back the histogram:
    unsigned int h_histogram1[10];
 
    hipMemcpy ((void *) h_histogram1,
                (void *) d_histogram1,
                10 * sizeof(unsigned int),
                hipMemcpyDeviceToHost);
    
    // make sure the GPU is finished doing everything!
    hipDeviceSynchronize();

    // how much time has elapsed?
    gettimeofday (&t1, 0);
 
    hipFree (d_histogram1);


    GPU_CHECKERROR(
    hipMalloc ((void **) &d_histogram2, 10 * sizeof (unsigned int))
    );
    GPU_CHECKERROR(
        hipMemset ((void *) d_histogram2, 0, 10 * sizeof (unsigned int))
    );

    gettimeofday (&t2, 0);

    // launch the kernel (with shared memory):
    histogram_gpu2<<<dimGrid, dimBlock>>>
                                        (x,y,z,
                                        d_intAArray,
                                        d_histogram2);

    // get back the histogram:
    unsigned int h_histogram2[10];
 
    hipMemcpy ((void *) h_histogram2,
                (void *) d_histogram2,
                10 * sizeof(unsigned int),
                hipMemcpyDeviceToHost);

    // make sure the GPU is finished doing everything!
    hipDeviceSynchronize();

    // finish timing:
    gettimeofday (&t3, 0);

    // free up the memory:
    hipFree (d_histogram2);
    hipFree (d_intAArray);
    free (h_intAArray);
 
    // complete the timing:
    float timdiff1 = (1000000.0*(t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)) / 1000000.0;
    float timdiff2 = (1000000.0*(t3.tv_sec - t2.tv_sec) + (t3.tv_usec - t2.tv_usec)) / 1000000.0;

    printf ("%.2f\n", timdiff1);
    for(int i=0;i<10;i++){
        printf ("%d ", h_histogram1[i]);
    }
    printf ("\n");

    printf ("%.2f\n", timdiff2);
    for(int i=0;i<10;i++){
        printf ("%d ", h_histogram2[i]);
    }
    printf ("\n");
 
}